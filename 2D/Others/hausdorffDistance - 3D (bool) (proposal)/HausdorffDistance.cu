#include "hip/hip_runtime.h"
#include<stdio.h>
#include<Volume.h>
#include<HausdorffDistance.cuh>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""


typedef unsigned char uchar;
typedef unsigned int uint;


__device__ int finished; //global variable that contains a boolean which indicates when to stop the kernel processing

__constant__ __device__ int WIDTH, HEIGHT, DEPTH; //constant variables that contain the size of the volume



__global__ void dilate(const bool *IMG1, const bool *IMG2, const bool *img1Read, const bool *img2Read, 
	bool *img1Write, bool *img2Write){

	const int id = blockDim.x * blockIdx.x + threadIdx.x;
	#if !IS_3D
	const int x = id % WIDTH, y = id / WIDTH;
	#else
	const int x = id % WIDTH, y = (id/WIDTH) % HEIGHT, z = (id/WIDTH)/HEIGHT;
	#endif

	if (id < WIDTH*HEIGHT*DEPTH){


		if (img1Read[id]){
			if (x + 1 < WIDTH) img1Write[id + 1] = true;
			if (x - 1 >= 0) img1Write[id - 1] = true;
			if (y + 1 < HEIGHT) img1Write[id + WIDTH] = true;
			if (y - 1 >= 0) img1Write[id - WIDTH] = true;
			#if IS_3D //if working with 3d volumes, then the 3D part
			if (z + 1 < DEPTH) img1Write[id + WIDTH*HEIGHT] = true;
			if (z - 1 >=0) img1Write[id - WIDTH*HEIGHT] = true;
			#endif
			
			#if CHEBYSHEV
			//diagonals
			if (x + 1 < WIDTH && y - 1 >= 0) img1Write[id - WIDTH + 1] = true;
			if (x - 1 >= 0 && y - 1 >= 0) img1Write[id - WIDTH - 1] = true;
			if (x + 1 < WIDTH && y + 1 < HEIGHT) img1Write[id + WIDTH + 1] = true;
			if (x - 1 >= 0 && y + 1 < HEIGHT) img1Write[id + WIDTH - 1] = true;
			#if IS_3D //if working with 3d volumes, then the 3D part
			if (z + 1 < DEPTH && x + 1 < WIDTH && y - 1 >= 0) img1Write[id - WIDTH + 1 + WIDTH*HEIGHT] = true;
			if (z + 1 < DEPTH && x - 1 >= 0 && y - 1 >= 0) img1Write[id - WIDTH - 1 + WIDTH*HEIGHT] = true;
			if (z + 1 < DEPTH && x + 1 < WIDTH && y + 1 < HEIGHT) img1Write[id + WIDTH + 1 + WIDTH*HEIGHT] = true;
			if (z + 1 < DEPTH && x - 1 >= 0 && y + 1 < HEIGHT) img1Write[id + WIDTH - 1 + WIDTH*HEIGHT] = true;
			if (z - 1 >= 0 && x + 1 < WIDTH && y - 1 >= 0) img1Write[id - WIDTH + 1 - WIDTH*HEIGHT] = true;
			if (z - 1 >= 0 && x - 1 >= 0 && y - 1 >= 0) img1Write[id - WIDTH - 1 - WIDTH*HEIGHT] = true;
			if (z - 1 >= 0 && x + 1 < WIDTH && y + 1 < HEIGHT) img1Write[id + WIDTH + 1 - WIDTH*HEIGHT] = true;
			if (z - 1 >= 0 && x - 1 >= 0 && y + 1 < HEIGHT) img1Write[id + WIDTH - 1 - WIDTH*HEIGHT] = true;
			#endif
			#endif
		}


		if (img2Read[id]){
			if (x + 1 < WIDTH) img2Write[id + 1] = true;
			if (x - 1 >= 0) img2Write[id - 1] = true;
			if (y + 1 < HEIGHT) img2Write[id + WIDTH] = true;
			if (y - 1 >= 0) img2Write[id - WIDTH] = true;
			#if IS_3D //if working with 3d volumes, then the 3D part
			if (z + 1 < DEPTH) img2Write[id + WIDTH*HEIGHT] = true;
			if (z - 1 >= 0) img2Write[id - WIDTH*HEIGHT] = true;
			#endif

			#if CHEBYSHEV
			//diagonals
			if (x + 1 < WIDTH && y - 1 >= 0) img2Write[id - WIDTH + 1] = true;
			if (x - 1 >= 0 && y - 1 >= 0) img2Write[id - WIDTH - 1] = true;
			if (x + 1 < WIDTH && y + 1 < HEIGHT) img2Write[id + WIDTH + 1] = true;
			if (x - 1 >= 0 && y + 1 < HEIGHT) img2Write[id + WIDTH - 1] = true;
			#if IS_3D //if working with 3d volumes, then the 3D part
			if (z + 1 < DEPTH && x + 1 < WIDTH && y - 1 >= 0) img2Write[id - WIDTH + 1 + WIDTH*HEIGHT] = true;
			if (z + 1 < DEPTH && x - 1 >= 0 && y - 1 >= 0) img2Write[id - WIDTH - 1 + WIDTH*HEIGHT] = true;
			if (z + 1 < DEPTH && x + 1 < WIDTH && y + 1 < HEIGHT) img2Write[id + WIDTH + 1 + WIDTH*HEIGHT] = true;
			if (z + 1 < DEPTH && x - 1 >= 0 && y + 1 < HEIGHT) img2Write[id + WIDTH - 1 + WIDTH*HEIGHT] = true;
			if (z - 1 >= 0 && x + 1 < WIDTH && y - 1 >= 0) img2Write[id - WIDTH + 1 - WIDTH*HEIGHT] = true;
			if (z - 1 >= 0 && x - 1 >= 0 && y - 1 >= 0) img2Write[id - WIDTH - 1 - WIDTH*HEIGHT] = true;
			if (z - 1 >= 0 && x + 1 < WIDTH && y + 1 < HEIGHT) img2Write[id + WIDTH + 1 - WIDTH*HEIGHT] = true;
			if (z - 1 >= 0 && x - 1 >= 0 && y + 1 < HEIGHT) img2Write[id + WIDTH - 1 - WIDTH*HEIGHT] = true;
			#endif
			#endif
		}


		//this is an atomic and computed to the finished global variable, if image 1 contains all of image 2 and image 2 contains all pixels of
		//image 1 then finished is true
		atomicAnd(&finished, (img2Read[id] || !IMG1[id]) && (img1Read[id] || !IMG2[id]));
	}
}


int HausdorffDistance::computeDistance(Volume *img1, Volume *img2){

	const int height = (*img1).getHeight(), width = (*img1).getWidth(), depth = (*img1).getDepth();

	size_t size = width*height*depth*sizeof(bool);

	//getting details of your CUDA device
	hipDeviceProp_t props;
	hipGetDeviceProperties(&props, CUDA_DEVICE_INDEX); //device index = 0, you can change it if you have more CUDA devices
	const int threadsPerBlock = props.maxThreadsPerBlock/2;
	const int blocksPerGrid = (height*width*depth + threadsPerBlock - 1) / threadsPerBlock;


	//copying the dimensions to the GPU
	hipMemcpyToSymbolAsync(HIP_SYMBOL(WIDTH), &width, sizeof(width));
	hipMemcpyToSymbolAsync(HIP_SYMBOL(HEIGHT), &height, sizeof(height));
	hipMemcpyToSymbolAsync(HIP_SYMBOL(DEPTH), &depth, sizeof(depth));


	//allocating the input images on the GPU
	bool *d_img1, *d_img2;
	hipMalloc(&d_img1, size);
	hipMalloc(&d_img2, size);


	//copying the data to the allocated memory on the GPU
	hipMemcpyAsync(d_img1, (*img1).getVolume(), size, hipMemcpyHostToDevice);
	hipMemcpyAsync(d_img2, (*img2).getVolume(), size, hipMemcpyHostToDevice);


	//allocating the images that will be the processing ones
	bool *d_img1Write, *d_img1Read, *d_img2Write, *d_img2Read;
	hipMalloc(&d_img1Write, size); hipMalloc(&d_img1Read, size);
	hipMalloc(&d_img2Write, size); hipMalloc(&d_img2Read, size);


	//cloning the input images to these two image versions (write and read)
	hipMemcpyAsync(d_img1Read, d_img1, size, hipMemcpyDeviceToDevice);
	hipMemcpyAsync(d_img2Read, d_img2, size, hipMemcpyDeviceToDevice);
	hipMemcpyAsync(d_img1Write, d_img1, size, hipMemcpyDeviceToDevice);
	hipMemcpyAsync(d_img2Write, d_img2, size, hipMemcpyDeviceToDevice);



	//required variables to compute the distance
	int h_finished = false, t = true;
	int distance = -1;

	//where the magic happens
	while (!h_finished){
		//reset the bool variable that verifies if the processing ended
		hipMemcpyToSymbol(HIP_SYMBOL(finished), &t, sizeof(h_finished));


		//lauching the verify kernel, which verifies if the processing finished
		dilate << < blocksPerGrid, threadsPerBlock >> >(d_img1, d_img2, d_img1Read, d_img2Read, d_img1Write, d_img2Write);

		//hipDeviceSynchronize();

		//updating the imgRead (cloning imgWrite to imgRead)
		hipMemcpy(d_img1Read, d_img1Write, size, hipMemcpyDeviceToDevice);
		hipMemcpy(d_img2Read, d_img2Write, size, hipMemcpyDeviceToDevice);

		

		//copying the result back to host memory
		hipMemcpyFromSymbol(&h_finished, HIP_SYMBOL(finished), sizeof(h_finished));


		//incrementing the distance at each iteration
		distance++;
	}


	//freeing memory
	hipFree(d_img1); hipFree(d_img2);
	hipFree(d_img1Write); hipFree(d_img1Read);
	hipFree(d_img2Write); hipFree(d_img2Read);

	//resetting device
	hipDeviceReset();

	print(hipGetLastError(), "processing CUDA. Something may be wrong with your CUDA device.");

	return distance;

}

inline void HausdorffDistance::print(hipError_t error, char* msg){
	if (error != hipSuccess)
	{
		printf("Error on %s ", msg);
		fprintf(stderr, "Error code: %s!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
}
