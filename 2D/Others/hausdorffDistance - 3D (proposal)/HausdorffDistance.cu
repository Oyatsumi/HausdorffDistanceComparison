#include "hip/hip_runtime.h"
#include<stdio.h>
#include<Volume.h>
#include<HausdorffDistance.cuh>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""


typedef unsigned char uchar;
typedef unsigned int uint;


//global variable that indicates when to stop processing
__device__ uint finished;

//constant variables that contain the size of the volume
__constant__ __device__ int WIDTH, HEIGHT, DEPTH;

//3d texture declaration
texture<uchar, hipTextureType3D, hipReadModeElementType> img1Tex, img2Tex;
//3d surface declaration
surface<void, 3> img1Surf, img2Surf;




__global__ void dilate(){

	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	const int z = blockIdx.z * blockDim.z + threadIdx.z;

	if (x < WIDTH && y < HEIGHT && z < DEPTH){

		const uchar p1 = tex3D(img1Tex, x + 0.5f, y + 0.5f, z + 0.5f), p2 = tex3D(img2Tex, x + 0.5f, y + 0.5f, z + 0.5f);

		if (p1 != 1){

			uchar res = tex3D(img1Tex, x - 1 + 0.5f, y + 0.5f, z + 0.5f) |
				tex3D(img1Tex, x + 1 + 0.5f, y + 0.5f, z + 0.5f) |
				tex3D(img1Tex, x + 0.5f, y - 1 + 0.5f, z + 0.5f) |
				tex3D(img1Tex, x + 0.5f, y + 1 + 0.5f, z + 0.5f);
			#if IS_3D
			res |= text3D(img1Tex, x + 0.5f, y + 0.5f, z + 1 + 0.5f) |
				text3D(img1Tex, x + 0.5f, y + 0.5f, z - 1 + 0.5f);
			#endif
			#if CHEBYSHEV
			res |= tex3D(img1Tex, x + 1 + 0.5f, y + 1 + 0.5f, z + 0.5f) |
				tex3D(img1Tex, x - 1 + 0.5f, y + 1 + 0.5f, z + 0.5f) |
				tex3D(img1Tex, x + 1 + 0.5f, y - 1 + 0.5f, z + 0.5f) |
				tex3D(img1Tex, x - 1 + 0.5f, y - 1 + 0.5f, z + 0.5f);
			#if IS_3D
			res |= tex3D(img1Tex, x + 1 + 0.5f, y + 1 + 0.5f, z + 1 +0.5f) |
				tex3D(img1Tex, x - 1 + 0.5f, y + 1 + 0.5f, z + 1 + 0.5f) |
				tex3D(img1Tex, x + 1 + 0.5f, y - 1 + 0.5f, z + 1 + 0.5f) |
				tex3D(img1Tex, x - 1 + 0.5f, y - 1 + 0.5f, z + 1 + 0.5f) |
				tex3D(img1Tex, x + 1 + 0.5f, y + 1 + 0.5f, z - 1 + 0.5f) |
				tex3D(img1Tex, x - 1 + 0.5f, y + 1 + 0.5f, z - 1 + 0.5f) |
				tex3D(img1Tex, x + 1 + 0.5f, y - 1 + 0.5f, z - 1 + 0.5f) |
				tex3D(img1Tex, x - 1 + 0.5f, y - 1 + 0.5f, z - 1 + 0.5f);
			#endif
			#endif
			res = (res > 0) ? 2 : 0;

			surf3Dwrite(res, img1Surf, x * sizeof(uchar), y, z);
		}

		if (p2 != 1){
			uchar res2 = tex3D(img2Tex, x - 1 + 0.5f, y + 0.5f, z + 0.5f) |
				tex3D(img2Tex, x + 1 + 0.5f, y + 0.5f, z + 0.5f) |
				tex3D(img2Tex, x + 0.5f, y - 1 + 0.5f, z + 0.5f) |
				tex3D(img2Tex, x + 0.5f, y + 1 + 0.5f, z + 0.5f);
			#if IS_3D
			res2 |= text3D(img2Tex, x + 0.5f, y + 0.5f, z + 1 + 0.5f) |
				text3D(img2Tex, x + 0.5f, y + 0.5f, z - 1 + 0.5f);
			#endif
			#if CHEBYSHEV
			res2 |= tex3D(img2Tex, x + 1 + 0.5f, y + 1 + 0.5f, z + 0.5f) |
				tex3D(img2Tex, x - 1 + 0.5f, y + 1 + 0.5f, z + 0.5f) |
				tex3D(img2Tex, x + 1 + 0.5f, y - 1 + 0.5f, z + 0.5f) |
				tex3D(img2Tex, x - 1 + 0.5f, y - 1 + 0.5f, z + 0.5f);
			#if IS_3D
			res2 |= tex3D(img2Tex, x + 1 + 0.5f, y + 1 + 0.5f, z + 1 + 0.5f) |
				tex3D(img2Tex, x - 1 + 0.5f, y + 1 + 0.5f, z + 1 + 0.5f) |
				tex3D(img2Tex, x + 1 + 0.5f, y - 1 + 0.5f, z + 1 + 0.5f) |
				tex3D(img2Tex, x - 1 + 0.5f, y - 1 + 0.5f, z + 1 + 0.5f) |
				tex3D(img2Tex, x + 1 + 0.5f, y + 1 + 0.5f, z - 1 + 0.5f) |
				tex3D(img2Tex, x - 1 + 0.5f, y + 1 + 0.5f, z - 1 + 0.5f) |
				tex3D(img2Tex, x + 1 + 0.5f, y - 1 + 0.5f, z - 1 + 0.5f) |
				tex3D(img2Tex, x - 1 + 0.5f, y - 1 + 0.5f, z - 1 + 0.5f);
			#endif
			#endif
			res2 = (res2 > 0) ? 2 : 0;
			surf3Dwrite(res2, img2Surf, x * sizeof(uchar), y, z);
		}


		atomicAnd(&finished, (p2 > 0 && p1 > 0) || (p2 != 1 && p1 != 1));

	}

}


int HausdorffDistance::computeDistance(Volume *img1, Volume *img2){

	const uint height = (*img1).getHeight(), width = (*img1).getWidth(), depth = (*img1).getDepth();

	hipArray *d_img1Array = 0, *d_img2Array = 0;
	const hipExtent volumeSize = make_hipExtent(width, height, depth);

	//copying the dimensions to the GPU
	hipMemcpyToSymbolAsync(HIP_SYMBOL(WIDTH), &width, sizeof(width));
	hipMemcpyToSymbolAsync(HIP_SYMBOL(HEIGHT), &height, sizeof(height));
	hipMemcpyToSymbolAsync(HIP_SYMBOL(DEPTH), &depth, sizeof(depth));

	// create 3D arrays
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<uchar>();
	//hipChannelFormatDesc channelDesc = hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindUnsigned);
	hipMalloc3DArray(&d_img1Array, &channelDesc, volumeSize);
	hipMalloc3DArray(&d_img2Array, &channelDesc, volumeSize);

	hipArray *d_img1SurfArray = 0, *d_img2SurfArray = 0;
	hipMalloc3DArray(&d_img1SurfArray, &channelDesc, volumeSize, hipArraySurfaceLoadStore);
	hipMalloc3DArray(&d_img2SurfArray, &channelDesc, volumeSize, hipArraySurfaceLoadStore);

	// copy data to 3D array
	hipMemcpy3DParms copyParams = { 0 };
	copyParams.srcPtr = make_hipPitchedPtr((void *)(*img1).getVolume(), volumeSize.width*sizeof(uchar), volumeSize.width, volumeSize.height);
	copyParams.extent = volumeSize;
	copyParams.kind = hipMemcpyHostToDevice;
	copyParams.dstArray = d_img1Array;
	print(hipMemcpy3D(&copyParams), "copying for img1 (texture).");
	//img2
	copyParams.srcPtr = make_hipPitchedPtr((void *)(*img2).getVolume(), volumeSize.width*sizeof(uchar), volumeSize.width, volumeSize.height);
	copyParams.dstArray = d_img2Array;
	print(hipMemcpy3D(&copyParams), "copying for img2 (texture).");


	//Copy data to 3D surfaces
	copyParams = { 0 };
	copyParams.extent = volumeSize;
	copyParams.kind = hipMemcpyDeviceToDevice;
	copyParams.dstArray = d_img1SurfArray;
	copyParams.srcArray = d_img1Array;
	print(hipMemcpy3D(&copyParams), "copying for surface1.");
	//img2
	//copyParams = { 0 };
	//copyParams.extent = volumeSize;
	//copyParams.kind = hipMemcpyDeviceToDevice;
	copyParams.dstArray = d_img2SurfArray;
	copyParams.srcArray = d_img2Array;
	print(hipMemcpy3D(&copyParams), "copying for surface2.");



	// set texture parameters
	img1Tex.normalized = false;                      // access with normalized texture coordinates
	//img1.filterMode = hipFilterModeLinear;      // linear interpolation
	img1Tex.addressMode[0] = hipAddressModeBorder;   // wrap texture coordinates
	img1Tex.addressMode[1] = hipAddressModeBorder;
	img1Tex.addressMode[2] = hipAddressModeBorder;
	//img2
	img2Tex.normalized = false;                      // access with normalized texture coordinates
	//img1.filterMode = hipFilterModeLinear;      // linear interpolation
	img2Tex.addressMode[0] = hipAddressModeBorder;   // wrap texture coordinates
	img2Tex.addressMode[1] = hipAddressModeBorder;
	img2Tex.addressMode[2] = hipAddressModeBorder;


	// bind array to 3D texture
	print(hipBindTextureToArray(img1Tex, d_img1Array, channelDesc), "binding texture to array 1.");
	print(hipBindTextureToArray(img2Tex, d_img2Array, channelDesc), "binding texture to array 2.");
	//bind array to 3D surface
	print(cudaBindSurfaceToArray(img1Surf, d_img1SurfArray, channelDesc), "binding surface to array 1.");
	print(cudaBindSurfaceToArray(img2Surf, d_img2SurfArray, channelDesc), "binding surface to array 2.");



	// launching kernel
	const int tSize = 8; //otimizar o tam automaticamente dps
	const dim3 threadsPerBlock(tSize, tSize, (*img1).getDepth() == 1 ? 1 : tSize);
	const dim3 blocksPerGrid((int)(width + threadsPerBlock.x - 1) / threadsPerBlock.x,
		(int)(height + threadsPerBlock.y - 1) / threadsPerBlock.y,
		(int)(depth + threadsPerBlock.z - 1) / threadsPerBlock.z);

	int distance = -1;
	uint loopEnded = false;
	const uint t = 1;
	//hipMalloc(&finished, sizeof(uint));
	while (!loopEnded){

		hipMemcpyToSymbolAsync(HIP_SYMBOL(finished), &t, sizeof(uint), 0, hipMemcpyHostToDevice);

		dilate << <blocksPerGrid, threadsPerBlock >> >();
		
		copyParams = { 0 };
		copyParams.extent = volumeSize;
		copyParams.kind = hipMemcpyDeviceToDevice;
		copyParams.srcArray = d_img1SurfArray;
		copyParams.dstArray = d_img1Array;
		hipMemcpy3D(&copyParams);

		//copyParams = { 0 };
		//copyParams.extent = volumeSize;
		//copyParams.kind = hipMemcpyDeviceToDevice;
		copyParams.srcArray = d_img2SurfArray;
		copyParams.dstArray = d_img2Array;
		hipMemcpy3D(&copyParams);

		hipMemcpyFromSymbol(&loopEnded, HIP_SYMBOL(finished), sizeof(uint), 0, hipMemcpyDeviceToHost);

		distance++;

	}

	print(hipGetLastError(), "kernel launch.");


	//free
	hipFreeArray(d_img1Array);
	hipFreeArray(d_img2Array);
	hipFreeArray(d_img1SurfArray);
	hipFreeArray(d_img2SurfArray);
	
	print(hipDeviceReset(), "device reset.");

	print(hipGetLastError(), "processing CUDA. Something may be wrong with your CUDA device.");

	return distance;
}

inline void HausdorffDistance::print(hipError_t error, char* msg){
	if (error != hipSuccess)
	{
		printf("Error on %s ", msg);
		fprintf(stderr, "Error code: %s!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
}
