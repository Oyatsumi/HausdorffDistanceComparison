#include "hip/hip_runtime.h"
#include<stdio.h>
#include<Volume.h>
#include<HausdorffDistance.cuh>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""


typedef unsigned char uchar;
typedef unsigned int uint;
typedef Volume::voxelStr voxelStrt;




__global__ void directedDistance(const voxelStrt *img1, const voxelStrt *img2, int *cMax, const int numVoxels1, const int numVoxels2){
	const int id = blockDim.x * blockIdx.x + threadIdx.x;
	
	
	//__shared__ int cmax;
	//if (threadIdx.x == 0) cmax = 0;
	//__syncthreads();

	int dist, cmin = 999999999;


	if (id < numVoxels1){

		const int x = img1[id].x, y = img1[id].y, z = img1[id].z;

		//computing the min distance
		for (int k = 0; k < numVoxels2; k++){
			//sup metric
			dist = abs(img2[k].x - x);
			dist = abs(img2[k].y - y) > dist ? abs(img2[k].y - y) : dist;
			dist = abs(img2[k].z - z) > dist ? abs(img2[k].z - z) : dist;
			//early break
			if (dist < (*cMax)) return; //early break
			//else, update
			cmin = (cmin > dist) ? dist : cmin;

		}
		atomicMax(cMax, cmin);
		//atomicMax(&cmax, cmin);
	}

	//__syncthreads();

	//if (threadIdx.x == 0){
	//	atomicMax(cMax, cmax);
	//}
	
}


int HausdorffDistance::computeDistance(Volume *img1, Volume *img2, Volume *img1D2, Volume *img2D2){

	//const int height = (*img1).getHeight(), width = (*img1).getWidth(), depth = (*img1).getDepth();

	size_t size1 = (*img1).getNumOfVoxels()*sizeof(voxelStrt), size2 = (*img2).getNumOfVoxels()*sizeof(voxelStrt),
		size1D2 = (*img1D2).getNumOfVoxels()*sizeof(voxelStrt), size2D2 = (*img2D2).getNumOfVoxels()*sizeof(voxelStrt);

	//getting details of your CUDA device
	hipDeviceProp_t props;
	hipGetDeviceProperties(&props, CUDA_DEVICE_INDEX); //change to the proper index of your cuda device
	const int threadsPerBlock = props.maxThreadsPerBlock/2;
	int blocksPerGrid = ((*img1).getNumOfVoxels() + threadsPerBlock - 1) / threadsPerBlock;

	

	//allocating the input data in the GPU
	voxelStrt *d_img1, *d_img2;
	hipMalloc(&d_img1, size1);
	hipMalloc(&d_img2, size2);
	int *d_cMax;
	hipMalloc(&d_cMax, sizeof(int));


	//copying the data to the allocated memory on the GPU
	hipMemcpy(d_img1, (*img1).getVolume(), size1, hipMemcpyHostToDevice);
	hipMemcpy(d_img2, (*img2).getVolume(), size2, hipMemcpyHostToDevice);

	//resetting cMax
	int t = 0;
	//hipMemcpyToSymbol(HIP_SYMBOL(cMax), &t, sizeof(t));
	hipMemcpy(d_cMax, &t, sizeof(int), hipMemcpyHostToDevice);

	//print(hipGetLastError(), "b1");

	//computing h(A,B)
	directedDistance << < blocksPerGrid, threadsPerBlock >> >(d_img1, d_img2, d_cMax, (*img1).getNumOfVoxels(), (*img2).getNumOfVoxels());

	//copying the data to the allocated memory on the GPU to compute the other directed distance
	hipDeviceSynchronize();
	hipFree(d_img1); hipFree(d_img2);
	hipMalloc(&d_img1, size1D2);
	hipMalloc(&d_img2, size2D2);
	hipMemcpy(d_img1, (*img1D2).getVolume(), size1D2, hipMemcpyHostToDevice);
	hipMemcpy(d_img2, (*img2D2).getVolume(), size2D2, hipMemcpyHostToDevice);

	//hipDeviceSynchronize();
	//print(hipGetLastError(), "b2");

	//resetting the variable
	blocksPerGrid = ((*img2).getNumOfVoxels() + threadsPerBlock - 1) / threadsPerBlock;

	//computing h(B,A)
	directedDistance << < blocksPerGrid, threadsPerBlock >> >(d_img2, d_img1, d_cMax, (*img2D2).getNumOfVoxels(), (*img1D2).getNumOfVoxels());
	
	hipDeviceSynchronize();
	//print(hipGetLastError(), "b3");

	//copying the result back
	int distance;
	//hipMemcpyFromSymbol(&distance, HIP_SYMBOL(cMax), sizeof(int));
	hipMemcpy(&distance, d_cMax, sizeof(int), hipMemcpyDeviceToHost);

	//print(hipGetLastError(), "b4");
	
	//freeing memory
	hipFree(d_img1); hipFree(d_img2);
	hipFree(d_cMax);

	
	//resetting device
	hipDeviceReset();

	print(hipGetLastError(), "processing CUDA. Something may be wrong with your CUDA device.");

	return distance;
}

inline void HausdorffDistance::print(hipError_t error, char* msg){
	if (error != hipSuccess)
	{
		printf("Error on %s ", msg);
		fprintf(stderr, "Error code: %s!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
}
