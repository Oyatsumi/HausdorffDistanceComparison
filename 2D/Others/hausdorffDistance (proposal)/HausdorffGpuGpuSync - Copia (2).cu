#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>




// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

//to call from .cpp
#include "Hausdorff_common.h"


//BLOCK SYNC
/*
__device__ volatile static int* arrayIn;
__device__ volatile static int* arrayOut;
__device__ void blockSync(int goalVal, volatile int *arrayIn, volatile int *arrayOut){
//thread ID in a block
int tId = threadIdx.x,
bNum = gridDim.x,
bId = blockIdx.x;

//only thread 0 is used for synchronization
if (tId == 0)
arrayIn[bId] = goalVal;

if (bId == 1){
if (tId < bNum){
while (arrayIn[tId] != goalVal){
//
}
}

__syncthreads();
if (tId < bNum){
arrayOut[tId] = goalVal;
}
}

if (tId == 0){
while (arrayOut[bId] != goalVal){
//
}
}
__syncthreads();
}
*/


__device__ int g_mutex = 0;

__device__ void blockSync2(int goalVal){
	//int tId = threadIdx.x;
	//if (tId == 0){
		atomicAdd(&g_mutex, 1);

		while (g_mutex != goalVal){
			//printf("gmuted: %d", g_mutex);
			//__syncthreads();
		}
	//}
	__syncthreads();
}
__device__ int g_mutex2 = 0;
__device__ void blockSync3(int goalVal){
	//int tId = threadIdx.x;
	//if (tId == 0){
		atomicAdd(&g_mutex2, 1);

		while (g_mutex2 != goalVal){
			//printf("gmuted: %d", g_mutex);
			//__syncthreads();
		}
	//}
	__syncthreads();
}

//N�O TAVA PEGANDO COMO RELEASE

__device__ static int grownEnough = false;
__device__ static int growReset = true;
//__device__ static int grownEnough = true;
__global__ void
hausdorffDistanceGPUSync(const bool *img1, const bool *img2,bool *img1P, bool *img2P, bool *img1PAux, bool *img2PAux,
const int WIDTH, const int HEIGHT, const int TILE_SIZE, const bool* structElement, const int STRUCT_SIZE, int *d_distance)
{
	extern __shared__ int imgsBuffer[];
	//if (threadIdx.x == 0) imgsBuffer[0] = 0;
	//bool* img1Buffer = &imgsBuffer[0];
	//bool* img2Buffer = &imgsBuffer[WIDTH*2+2];
	//__shared__ long bla[10000]; //enchendo a memoria
	const int id = blockDim.x * blockIdx.x + threadIdx.x;

	//__shared__ int grownEnoughBlock = true;
	//if (threadIdx.x == 0) grownEnoughBlock = 1;
	//__syncthreads();

	/*
	if (threadIdx.x == 0){
	img1b[]
	}*/
	/*
	//populate buffer
	for (int k = 0; k < TILE_SIZE; k++){//for tilesize
		currentId = id*TILE_SIZE + k;
		img1Buffer[k] = img1P[currentId];
		img2Buffer[k] = img2P[currentId];
	}*/

	int dist = 0, currentId = 0;
	while (!grownEnough || dist == 0 || g_mutex != blockDim.x*gridDim.x /*esse ultimo foi extremamente necessario*/){
		//printf("bla \n");
		//if (id == 0) printf(".");
		//reset grownEnough
		if (id == 0) atomicOr(&grownEnough, true);
		//updating imgP
		/*
		for (int k = 0; k < TILE_SIZE; k++){//for tilesize
			currentId = id*TILE_SIZE + k;
			img1P[currentId] = img1Buffer[k];
			img2P[currentId] = img2Buffer[k];
		}*/
		g_mutex = 0;
		blockSync3(blockDim.x*gridDim.x);
		//printf("%d", TILE_SIZE);
		for (int k = 0; k < TILE_SIZE; k++){//for tilesize
			currentId = id*TILE_SIZE + k;

			if (currentId < WIDTH*HEIGHT){
				//printf("[currentId: %d, img1: %d, img2: %d] ", currentId, img1P[currentId], img2P[currentId]);

				if (img1PAux[currentId]){
					if (currentId + 1 < WIDTH*HEIGHT) img1P[currentId + 1] = true;
					if (currentId - 1 >= 0) img1P[currentId - 1] = true;
					if (currentId + WIDTH < WIDTH*HEIGHT)img1P[currentId + WIDTH] = true;
					if (currentId - WIDTH >= 0) img1P[currentId - WIDTH] = true;
					//diagonais
					if (currentId - WIDTH + 1 >= 0) img1P[currentId - WIDTH + 1] = true;
					if (currentId - WIDTH - 1 >= 0) img1P[currentId - WIDTH - 1] = true;
					if (currentId + WIDTH + 1 < WIDTH*HEIGHT) img1P[currentId + WIDTH + 1] = true;
					if (currentId + WIDTH - 1 < WIDTH*HEIGHT) img1P[currentId + WIDTH - 1] = true;
				}
				if (img2PAux[currentId]){
					if (currentId + 1 < WIDTH*HEIGHT) img2P[currentId + 1] = true;
					if (currentId - 1 >= 0) img2P[currentId - 1] = true;
					if (currentId + WIDTH < WIDTH*HEIGHT) img2P[currentId + WIDTH] = true;
					if (currentId - WIDTH >= 0) img2P[currentId - WIDTH] = true;
					//diagonais
					if (currentId - WIDTH + 1 >= 0) img2P[currentId - WIDTH + 1] = true;
					if (currentId - WIDTH - 1 >= 0) img2P[currentId - WIDTH - 1] = true;
					if (currentId + WIDTH + 1 < WIDTH*HEIGHT) img2P[currentId + WIDTH + 1] = true;
					if (currentId + WIDTH - 1 < WIDTH*HEIGHT) img2P[currentId + WIDTH - 1] = true;
				}
				//hasGrownEnough(currentId, img1, img2, img1P, img2P, WIDTH, HEIGHT, &grownEnoughBlock);
				//__syncthreads();
				//if (threadcurrentIdx.x == 0)
				atomicAnd(&grownEnough, (img2PAux[currentId] || !img1[currentId]) && (img1PAux[currentId] || !img2[currentId]));
				//grownEnough &= (img2P[currentId] || !img1[currentId]) && (img1P[currentId] || !img2[currentId]);
				//if (currentId == 0) finished = &grownEnough;
				//if (currentId == 0) printf("\n finished %d", *grownEnough);
			}
		}
		dist++;
		//blockSync(blockDim.x*dist, arrayIn, arrayOut);
		if (id == 0) atomicOr(&growReset, true);
		g_mutex2 = 0;
		blockSync2(blockDim.x*gridDim.x);
		//COPIAR DA SHARED PRA IMG1P e IMG2p
		for (int k = 0; k < TILE_SIZE; k++){//for tilesize
			currentId = id*TILE_SIZE + k;
			if (currentId < WIDTH*HEIGHT){
				img1PAux[currentId] = img1P[currentId];
				img2PAux[currentId] = img2P[currentId];
			}
			//atomicOr(&img1PAux[currentId], img1P[currentId]);
			//atomicOr(&img2PAux[currentId], img2P[currentId]);
		}
		//__threadfence();
		//if (id == 0) printf(".");
	}
	//if (id == 0) printf("terminou %d\n", dist);
	*d_distance = dist;
}






/**
* Host main routine
*/


int
hdGPUSync(bool *img1, bool *img2, const int WIDTH, const int HEIGHT, bool *structElement, const int STRUCT_SIZE)
{
	// Error code to check return values for CUDA calls
	hipError_t err = hipSuccess;

	// Print the vector length to be used, and compute its size
	//int numElements = 50000;
	//size_t size = numElements * sizeof(float);
	printf("Processing images (width=%d, height=%d)...\n", WIDTH, HEIGHT);

	// Allocate the host input vector A
	//float *h_A = (float *)malloc(size);

	// Allocate the host input vector B
	//float *h_B = (float *)malloc(size);

	// Allocate the host output vector C
	//float *h_C = (float *)malloc(size);

	// Verify that allocations succeeded
	/*
	if (h_A == NULL || h_B == NULL || h_C == NULL)
	{
	fprintf(stderr, "Failed to allocate host vectors!\n");
	exit(EXIT_FAILURE);
	}

	// Initialize the host input vectors
	for (int i = 0; i < numElements; ++i)
	{
	h_A[i] = rand() / (float)RAND_MAX;
	h_B[i] = rand() / (float)RAND_MAX;
	}*/

	size_t size = WIDTH*HEIGHT*sizeof(bool);
	//short *h_distance = (short *)malloc(sizeof(short));

	//Kernel variables
	//int threadsPerBlock = 512;
	//int blocksPerGrid = (WIDTH*HEIGHT + threadsPerBlock - 1) / threadsPerBlock;
	int blocksPerGrid = 5;
	int threadsPerBlock = 1024;
	int TILE_SIZE = (WIDTH*HEIGHT + threadsPerBlock*blocksPerGrid - 1) / (threadsPerBlock*blocksPerGrid);

	// Allocate the device input vector img1
	bool *d_img1 = NULL;
	err = hipMalloc((void **)&d_img1, size);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector d_img1 (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Allocate the device input vector img2
	bool *d_img2 = NULL;
	err = hipMalloc((void **)&d_img2, size);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector d_img2 (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Allocate the device input vector img1P
	bool *d_img1P = NULL;
	err = hipMalloc((void **)&d_img1P, size);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector d_img1P (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Allocate the device input vector img2P
	bool *d_img2P = NULL;
	err = hipMalloc((void **)&d_img2P, size);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector d_img2P (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Allocate the device input vector img2P
	bool *d_img1PAux = NULL;
	err = hipMalloc((void **)&d_img1PAux, size);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector d_img1PAux (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Allocate the device input vector img2P
	bool *d_img2PAux = NULL;
	err = hipMalloc((void **)&d_img2PAux, size);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector d_img2PAux (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}


	// Allocate the device structElement
	size_t STRUCT_SIZE_T = (STRUCT_SIZE + 1)*(STRUCT_SIZE + 1)*sizeof(bool);
	bool *d_structElement = NULL;
	err = hipMalloc((void **)&d_structElement, STRUCT_SIZE_T);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector d_structElement (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}




	// Allocate the device output grownEnough var
	int *d_distance = NULL;
	err = hipMalloc((void **)&d_distance, sizeof(int));

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device d_distance (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Copy the host input vectors A and B in host memory to the device input vectors in
	// device memory
	printf("Copy input data from the host memory to the CUDA device\n");
	err = hipMemcpy(d_img1, img1, size, hipMemcpyHostToDevice);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector img1 from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_img2, img2, size, hipMemcpyHostToDevice);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector img2 from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_img1P, d_img1, size, hipMemcpyDeviceToDevice);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector img1P from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_img2P, d_img2, size, hipMemcpyDeviceToDevice);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector img2P from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_img1PAux, d_img1, size, hipMemcpyDeviceToDevice);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector img1PAux from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_img2PAux, d_img2, size, hipMemcpyDeviceToDevice);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector img2PAux from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_structElement, structElement, STRUCT_SIZE_T, hipMemcpyHostToDevice);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector structElement from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}


	int distance = 0;

	err = hipMemcpy(d_distance, &distance, sizeof(int), hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector d_distance from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	//possivelmente device mem copy struct...
	printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
	hausdorffDistanceGPUSync << <blocksPerGrid, threadsPerBlock, 12288*sizeof(int) >> >
		(d_img1, d_img2, d_img1P, d_img2P, d_img1PAux, d_img2PAux, WIDTH, HEIGHT, TILE_SIZE, d_structElement, STRUCT_SIZE, d_distance);
	hipDeviceSynchronize();
	err = hipGetLastError();
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to launch hausdorffDistance kernel (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(&distance, d_distance, sizeof(int), hipMemcpyDeviceToHost);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector d_distance from device to host (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	printf("Hausdorff distance: %d\n", distance);




	/*
	// Verify that the result vector is correct
	for (int i = 0; i < numElements; ++i)
	{
	if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
	{
	fprintf(stderr, "Result verification failed at element %d!\n", i);
	exit(EXIT_FAILURE);
	}
	}

	printf("Test PASSED\n");
	*/

	// Free device global memory
	err = hipFree(d_img1);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to free device vector img1 (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipFree(d_img2);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to free device vector img2 (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipFree(d_img1P);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to free device img1P (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipFree(d_img2P);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to free device img2P (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}


	err = hipFree(d_img1PAux);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to free device img1PAux (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipFree(d_img2PAux);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to free device img2PAux (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}





	err = hipFree(d_structElement);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to free d_structElement (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipFree(d_distance);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to free d_structElement (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}




	// Free host memory
	//free(h_distance);
	/*
	free(h_A);
	free(h_B);
	free(h_C);
	*/

	// Reset the device and exit
	// hipDeviceReset causes the driver to clean up all state. While
	// not mandatory in normal operation, it is good practice.  It is also
	// needed to ensure correct operation when the application is being
	// profiled. Calling hipDeviceReset causes all profile data to be
	// flushed before the application exits
	err = hipDeviceReset();

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	printf("Done\n");
	return 0;
}




