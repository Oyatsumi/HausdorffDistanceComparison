#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>




// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

//to call from .cpp
#include "Hausdorff_common.h"


//BLOCK SYNC

__device__ volatile static int* arrayIn;
__device__ volatile static int* arrayOut;
__device__ void blockSync(int goalVal, volatile int *arrayIn, volatile int *arrayOut){
	//thread ID in a block
	int tId = threadIdx.x,
		bNum = gridDim.x,
		bId = blockIdx.x;

	//only thread 0 is used for synchronization
	if (tId == 0)
		arrayIn[bId] = goalVal;

	if (bId == 1){
		if (tId < bNum){
			while (arrayIn[tId] != goalVal){
				//
			}
		}

		__syncthreads();
		if (tId < bNum){
			arrayOut[tId] = goalVal;
		}
	}

	if (tId == 0){
		while (arrayOut[bId] != goalVal){
			//
		}
	}
	__syncthreads();
}



__device__ volatile static int g_mutex = 0;

__device__ void blockSync2(int goalVal){
	int tId = threadIdx.x;
	if (tId == 0){
		atomicAdd((int *)&g_mutex, 1);

		while (g_mutex != goalVal){
			//printf("gmuted: %d", g_mutex);
		}
	}
	__syncthreads();
}
__device__ volatile static int g_mutex2 = 0;
__device__ void blockSync3(int goalVal){
	int tId = threadIdx.x;
	if (tId == 0){
		atomicAdd((int *)&g_mutex2, 1);

		while (g_mutex2 != goalVal){
			//printf("gmuted: %d", g_mutex);
		}
	}
	__syncthreads();
}

__device__ static int grownEnough = true;
__device__ static int growReset = true;
//__device__ static int grownEnough = true;
__global__ void
hausdorffDistanceGPUSync(const bool *img1, const bool *img2, volatile bool *img1P, volatile bool *img2P, 
const int WIDTH, const int HEIGHT, const bool* structElement, const char STRUCT_SIZE, int *d_distance)
{
	extern __shared__ int img2Buffer[];
	if (threadIdx.x == 0) img2Buffer[0] = 0;
	//__shared__ long bla[10000]; //enchendo a memoria
	const int id = blockDim.x * blockIdx.x + threadIdx.x;

	//__shared__ int grownEnoughBlock = true;
	//if (threadIdx.x == 0) grownEnoughBlock = 1;
	//__syncthreads();

	/*
	if (threadIdx.x == 0){
	img1b[]
	}*/
	int dist = 0;
	while (!grownEnough || dist == 0){
		printf("bla \n");
		/*
		if (g_mutex == (gridDim.x)) {
			g_mutex = 0; //resetting
			grownEnough = true;
			__syncthreads();
		}*/
		//grownEnough = true;
		//printf("teste: %d, %d", g_mutex, gridDim.x*dist);
		/*
		if (growReset && threadIdx.x == 0) {//devia ser sync de bloco
			atomicOr(&grownEnough, true);
			atomicAnd(&growReset, false);
		}
		__syncthreads();*/
		if (id == 0) atomicOr(&grownEnough, true);
		blockSync3(gridDim.x*(dist+1));
		if (id < WIDTH*HEIGHT){
			//printf("[id: %d, img1: %d, img2: %d] ", id, img1P[id], img2P[id]);

			if (img1P[id]){
				if (id + 1 < WIDTH*HEIGHT) img1P[id + 1] = true;
				if (id - 1 >= 0) img1P[id - 1] = true;
				if (id + WIDTH < WIDTH*HEIGHT)img1P[id + WIDTH] = true;
				if (id - WIDTH >= 0) img1P[id - WIDTH] = true;
				//diagonais
				if (id - WIDTH + 1 >= 0) img1P[id - WIDTH + 1] = true;
				if (id - WIDTH - 1 >= 0) img1P[id - WIDTH - 1] = true;
				if (id + WIDTH + 1 < WIDTH*HEIGHT) img1P[id + WIDTH + 1] = true;
				if (id + WIDTH - 1 < WIDTH*HEIGHT) img1P[id + WIDTH - 1] = true;
			}
			if (img2P[id]){
				if (id + 1 < WIDTH*HEIGHT) img2P[id + 1] = true;
				if (id - 1 >= 0) img2P[id - 1] = true;
				if (id + WIDTH < WIDTH*HEIGHT) img2P[id + WIDTH] = true;
				if (id - WIDTH >= 0) img2P[id - WIDTH] = true;
				//diagonais
				if (id - WIDTH + 1 >= 0) img2P[id - WIDTH + 1] = true;
				if (id - WIDTH - 1 >= 0) img2P[id - WIDTH - 1] = true;
				if (id + WIDTH + 1 < WIDTH*HEIGHT) img2P[id + WIDTH + 1] = true;
				if (id + WIDTH - 1 < WIDTH*HEIGHT) img2P[id + WIDTH - 1] = true;
			}

			/*
			//pixel index (linear)
			int pIndex;
			//divergencia de dados - precisa de sincroniza��o
			for (char dY = -1; dY <= 1; dY++){
			for (char dX = -1; dX <= 1; dX++){
			if (dX == 0 || dY == 0) continue;
			for (char i = 0; i <= STRUCT_SIZE; i++){//structuring element
			for (char j = 0; j <= STRUCT_SIZE; j++){
			//if (id == 0)printf("\n bla0 %d, %d\n", (STRUCT_SIZE + 1)*i + j, structElement[(STRUCT_SIZE + 1)*i + j]);
			if (!structElement[(STRUCT_SIZE+1)*i + j]) continue;
			//if (id==0)printf("\n bla i:%d, j:%d, dX:%d, dY:%d \n", i*dX, j*dY, dX, dY);

			pIndex = id + (j*dX) + WIDTH*(i*dY);
			//if (id==0)printf("[he: %d \n", pIndex);
			if (img1[id]){//if there is a pixel in img1, dilate
			if (pIndex >= 0 && pIndex < WIDTH*HEIGHT){//boundaries
			img1P[pIndex] = true;
			}
			}
			if (img2[id]){//if there is a pixel in img2, dilate
			if (pIndex >= 0 && pIndex < WIDTH*HEIGHT){//boundaries
			img2P[pIndex] = true;
			}
			}
			}
			}
			}
			}
			*/
			//printf("[id: %d, img1: %d, img2: %d] ", id, img1P[id], img2P[id]);
			//hasGrownEnough(id, img1, img2, img1P, img2P, WIDTH, HEIGHT, &grownEnoughBlock);
			//__syncthreads();
			//if (threadIdx.x == 0)
			atomicAnd(&grownEnough, (img2P[id] || !img1[id]) && (img1P[id] || !img2[id]));
			//grownEnough &= (img2P[id] || !img1[id]) && (img1P[id] || !img2[id]);
			//if (id == 0) finished = &grownEnough;
			//if (id == 0) printf("\n finished %d", *grownEnough);
		}
		dist++;
		//blockSync(blockDim.x*dist, arrayIn, arrayOut);
		if (id == 0) atomicOr(&growReset, true);
		blockSync2(gridDim.x*dist);
		//
		if (id == 0) printf("dps do sync");
	}
	if (id == 0) printf("terminou %d\n", dist);
	*d_distance = dist;
}






/**
* Host main routine
*/


int
hdGPUSync(bool *img1, bool *img2, const int WIDTH, const int HEIGHT, bool *structElement, const int STRUCT_SIZE)
{
	// Error code to check return values for CUDA calls
	hipError_t err = hipSuccess;

	// Print the vector length to be used, and compute its size
	//int numElements = 50000;
	//size_t size = numElements * sizeof(float);
	printf("Processing images (width=%d, height=%d)...\n", WIDTH, HEIGHT);

	// Allocate the host input vector A
	//float *h_A = (float *)malloc(size);

	// Allocate the host input vector B
	//float *h_B = (float *)malloc(size);

	// Allocate the host output vector C
	//float *h_C = (float *)malloc(size);

	// Verify that allocations succeeded
	/*
	if (h_A == NULL || h_B == NULL || h_C == NULL)
	{
	fprintf(stderr, "Failed to allocate host vectors!\n");
	exit(EXIT_FAILURE);
	}

	// Initialize the host input vectors
	for (int i = 0; i < numElements; ++i)
	{
	h_A[i] = rand() / (float)RAND_MAX;
	h_B[i] = rand() / (float)RAND_MAX;
	}*/

	size_t size = WIDTH*HEIGHT*sizeof(bool);
	//short *h_distance = (short *)malloc(sizeof(short));

	//Kernel variables
	//int threadsPerBlock = 512;
	//int blocksPerGrid = (WIDTH*HEIGHT + threadsPerBlock - 1) / threadsPerBlock;
	int blocksPerGrid = 5;
	int threadsPerBlock = 1024;

	// Allocate the device input vector img1
	bool *d_img1 = NULL;
	err = hipMalloc((void **)&d_img1, size);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector d_img1 (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Allocate the device input vector img2
	bool *d_img2 = NULL;
	err = hipMalloc((void **)&d_img2, size);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector d_img2 (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Allocate the device input vector img1P
	bool *d_img1P = NULL;
	err = hipMalloc((void **)&d_img1P, WIDTH*HEIGHT*sizeof(volatile bool));

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector d_img1P (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Allocate the device input vector img2P
	bool *d_img2P = NULL;
	err = hipMalloc((void **)&d_img2P, WIDTH*HEIGHT*sizeof(volatile bool));

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector d_img2P (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}


	// Allocate the device structElement
	size_t STRUCT_SIZE_T = (STRUCT_SIZE + 1)*(STRUCT_SIZE + 1)*sizeof(bool);
	bool *d_structElement = NULL;
	err = hipMalloc((void **)&d_structElement, STRUCT_SIZE_T);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector d_structElement (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}




	// Allocate the device output grownEnough var
	int *d_distance = NULL;
	err = hipMalloc((void **)&d_distance, sizeof(int));

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device d_distance (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Copy the host input vectors A and B in host memory to the device input vectors in
	// device memory
	printf("Copy input data from the host memory to the CUDA device\n");
	err = hipMemcpy(d_img1, img1, size, hipMemcpyHostToDevice);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector img1 from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_img2, img2, size, hipMemcpyHostToDevice);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector img2 from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_img1P, img1, size, hipMemcpyHostToDevice);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector img1P from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_img2P, img2, size, hipMemcpyHostToDevice);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector img2P from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_structElement, structElement, STRUCT_SIZE_T, hipMemcpyHostToDevice);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector structElement from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}


	int distance = 0;

	err = hipMemcpy(d_distance, &distance, sizeof(int), hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector d_distance from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	//possivelmente device mem copy struct...
	printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
	hausdorffDistanceGPUSync << <blocksPerGrid, threadsPerBlock, 12288*sizeof(int)>> >
		(d_img1, d_img2, d_img1P, d_img2P, WIDTH, HEIGHT, d_structElement, STRUCT_SIZE, d_distance);
	err = hipGetLastError();
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to launch hausdorffDistance kernel (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(&distance, d_distance, sizeof(int), hipMemcpyDeviceToHost);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector d_distance from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	printf("Hausdorff distance: %d\n", distance);
	

	

	/*
	// Verify that the result vector is correct
	for (int i = 0; i < numElements; ++i)
	{
	if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
	{
	fprintf(stderr, "Result verification failed at element %d!\n", i);
	exit(EXIT_FAILURE);
	}
	}

	printf("Test PASSED\n");
	*/

	// Free device global memory
	err = hipFree(d_img1);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to free device vector img1 (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipFree(d_img2);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to free device vector img2 (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipFree(d_img1P);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to free device img1P (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipFree(d_img2P);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to free device img2P (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}




	err = hipFree(d_structElement);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to free d_structElement (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipFree(d_distance);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to free d_structElement (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}




	// Free host memory
	//free(h_distance);
	/*
	free(h_A);
	free(h_B);
	free(h_C);
	*/

	// Reset the device and exit
	// hipDeviceReset causes the driver to clean up all state. While
	// not mandatory in normal operation, it is good practice.  It is also
	// needed to ensure correct operation when the application is being
	// profiled. Calling hipDeviceReset causes all profile data to be
	// flushed before the application exits
	err = hipDeviceReset();

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	printf("Done\n");
	return 0;
}




