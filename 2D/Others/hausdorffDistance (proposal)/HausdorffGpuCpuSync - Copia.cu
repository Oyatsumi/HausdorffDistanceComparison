#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>




// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

//to call from .cpp
#include "Hausdorff_common.h"



__device__ void
hasGrownEnough(const int id, const bool *img1, const bool *img2, bool *img1P, bool *img2P, const int WIDTH, const int HEIGHT, int *grownEnough){
	//__shared__ bool grownEnoughLocal = true;
	//if (id >= 0 && id < WIDTH*HEIGHT){
		//bool coveredImg1 = img2P[id] || !img1[id],
		//	coveredImg2 = img1P[id] || !img2[id],
		//	covered = coveredImg1 && coveredImg2;
		//atomicAnd(grownEnough, covered);
		 //*grownEnough &= (img2P[id] || !img1[id]) && (img1P[id] || !img2[id]);
	//}
}


//__device__ static int grownEnough = true;
__global__ void
hausdorffDistanceCPUSync(const bool *img1, const bool *img2, bool *img1P, bool *img2P, int *grownEnough,
const int WIDTH, const int HEIGHT, const bool* structElement, const char STRUCT_SIZE)
{
	//const int id = blockDim.x * blockIdx.x + threadIdx.x;
	const int id = blockDim.x * blockIdx.x + threadIdx.x;

	//__shared__ int grownEnoughBlock;
	//if (threadIdx.x == 0) grownEnoughBlock = 1;
	//__syncthreads();

	/*
	if (threadIdx.x == 0){
		img1b[]
	}*/

	if (id < WIDTH*HEIGHT){
		//printf("[id: %d, img1: %d, img2: %d] ", id, img1P[id], img2P[id]);

		if (img1[id]){
			if (id + 1 < WIDTH*HEIGHT) img1P[id + 1] = true;
			if (id - 1 >= 0) img1P[id - 1] = true;
			if (id + WIDTH < WIDTH*HEIGHT)img1P[id + WIDTH] = true;
			if (id - WIDTH >= 0) img1P[id - WIDTH] = true;
			//diagonais
			if (id - WIDTH + 1 >= 0) img1P[id - WIDTH + 1] = true;
			if (id - WIDTH - 1 >= 0) img1P[id - WIDTH - 1] = true;
			if (id + WIDTH + 1 < WIDTH*HEIGHT) img1P[id + WIDTH + 1] = true;
			if (id + WIDTH - 1 < WIDTH*HEIGHT) img1P[id + WIDTH - 1] = true;
		}
		if (img2[id]){
			if (id + 1 < WIDTH*HEIGHT) img2P[id + 1] = true;
			if (id - 1 >= 0) img2P[id - 1] = true;
			if (id + WIDTH < WIDTH*HEIGHT) img2P[id + WIDTH] = true;
			if (id - WIDTH >= 0) img2P[id - WIDTH] = true;
			//diagonais
			if (id - WIDTH + 1 >= 0) img2P[id - WIDTH + 1] = true;
			if (id - WIDTH - 1 >= 0) img2P[id - WIDTH - 1] = true;
			if (id + WIDTH + 1 < WIDTH*HEIGHT) img2P[id + WIDTH + 1] = true;
			if (id + WIDTH - 1 < WIDTH*HEIGHT) img2P[id + WIDTH - 1] = true;
		}
		
		/*
		//pixel index (linear)
		int pIndex;
		//divergencia de dados - precisa de sincroniza��o
		for (char dY = -1; dY <= 1; dY++){
			for (char dX = -1; dX <= 1; dX++){
				if (dX == 0 || dY == 0) continue;
				for (char i = 0; i <= STRUCT_SIZE; i++){//structuring element
					for (char j = 0; j <= STRUCT_SIZE; j++){
						//if (id == 0)printf("\n bla0 %d, %d\n", (STRUCT_SIZE + 1)*i + j, structElement[(STRUCT_SIZE + 1)*i + j]);
						if (!structElement[(STRUCT_SIZE+1)*i + j]) continue;
						//if (id==0)printf("\n bla i:%d, j:%d, dX:%d, dY:%d \n", i*dX, j*dY, dX, dY);

						pIndex = id + (j*dX) + WIDTH*(i*dY);
						//if (id==0)printf("[he: %d \n", pIndex);
						if (img1[id]){//if there is a pixel in img1, dilate
							if (pIndex >= 0 && pIndex < WIDTH*HEIGHT){//boundaries
								img1P[pIndex] = true;
							}
						}
						if (img2[id]){//if there is a pixel in img2, dilate
							if (pIndex >= 0 && pIndex < WIDTH*HEIGHT){//boundaries
								img2P[pIndex] = true;
							}
						}
					}
				}
			}
		}
		*/
		//printf("[id: %d, img1: %d, img2: %d] ", id, img1P[id], img2P[id]);
		//hasGrownEnough(id, img1, img2, img1P, img2P, WIDTH, HEIGHT, &grownEnoughBlock);
		//__syncthreads();
		//if (threadIdx.x == 0)
			atomicAnd(grownEnough, (img2P[id] || !img1[id]) && (img1P[id] || !img2[id]));
		//if (id == 0) finished = &grownEnough;
		//if (id == 0) printf("\n finished %d", *grownEnough);
	}

}






/**
* Host main routine
*/


int
hdCPUSync(bool *img1, bool *img2, const int WIDTH, const int HEIGHT, bool *structElement, const int STRUCT_SIZE)
{
	// Error code to check return values for CUDA calls
	hipError_t err = hipSuccess;

	// Print the vector length to be used, and compute its size
	//int numElements = 50000;
	//size_t size = numElements * sizeof(float);
	printf("Processing images (width=%d, height=%d)...\n", WIDTH, HEIGHT);

	// Allocate the host input vector A
	//float *h_A = (float *)malloc(size);

	// Allocate the host input vector B
	//float *h_B = (float *)malloc(size);

	// Allocate the host output vector C
	//float *h_C = (float *)malloc(size);

	// Verify that allocations succeeded
	/*
	if (h_A == NULL || h_B == NULL || h_C == NULL)
	{
	fprintf(stderr, "Failed to allocate host vectors!\n");
	exit(EXIT_FAILURE);
	}

	// Initialize the host input vectors
	for (int i = 0; i < numElements; ++i)
	{
	h_A[i] = rand() / (float)RAND_MAX;
	h_B[i] = rand() / (float)RAND_MAX;
	}*/

	size_t size = WIDTH*HEIGHT*sizeof(bool);
	//short *h_distance = (short *)malloc(sizeof(short));

	//Kernel variables
	int threadsPerBlock = 512;
	int blocksPerGrid = (WIDTH*HEIGHT + threadsPerBlock - 1) / threadsPerBlock;

	// Allocate the device input vector img1
	bool *d_img1 = NULL;
	err = hipMalloc((void **)&d_img1, size);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector d_img1 (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Allocate the device input vector img2
	bool *d_img2 = NULL;
	err = hipMalloc((void **)&d_img2, size);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector d_img2 (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Allocate the device input vector img1P
	bool *d_img1P = NULL;
	err = hipMalloc((void **)&d_img1P, size);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector d_img1P (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Allocate the device input vector img2P
	bool *d_img2P = NULL;
	err = hipMalloc((void **)&d_img2P, size);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector d_img2P (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}


	// Allocate the device structElement
	size_t STRUCT_SIZE_T = (STRUCT_SIZE + 1)*(STRUCT_SIZE + 1)*sizeof(bool);
	bool *d_structElement = NULL;
	err = hipMalloc((void **)&d_structElement, STRUCT_SIZE_T);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector d_structElement (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}


	// Allocate the device output grownEnough var
	int *d_grownEnough = NULL;
	err = hipMalloc((void **)&d_grownEnough, sizeof(int));

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device d_grownEnough (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Copy the host input vectors A and B in host memory to the device input vectors in
	// device memory
	printf("Copy input data from the host memory to the CUDA device\n");
	err = hipMemcpyAsync(d_img1, img1, size, hipMemcpyHostToDevice);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector img1 from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMemcpyAsync(d_img2, img2, size, hipMemcpyHostToDevice);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector img2 from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMemcpyAsync(d_img1P, img1, size, hipMemcpyHostToDevice);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector img1P from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMemcpyAsync(d_img2P, img2, size, hipMemcpyHostToDevice);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector img2P from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMemcpyAsync(d_structElement, structElement, STRUCT_SIZE_T, hipMemcpyHostToDevice);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector structElement from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	hipDeviceSynchronize();

	int h_grownEnough = false;
	int distance = 0, aux = 1;
	// Launch the Vector Add CUDA Kernel
	printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
	while (!h_grownEnough){
	//while (distance < 1){
		err = hipMemcpy(d_grownEnough, &aux, sizeof(int), hipMemcpyHostToDevice);
		if (err != hipSuccess)
		{
			fprintf(stderr, "Failed to copy vector d_grownEnough from host to device (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		//printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
		//possivelmente device mem copy struct...
		hausdorffDistanceCPUSync << <blocksPerGrid, threadsPerBlock>> >
			(d_img1, d_img2, d_img1P, d_img2P, d_grownEnough, WIDTH, HEIGHT, d_structElement, STRUCT_SIZE);
		err = hipGetLastError();
		if (err != hipSuccess)
		{
			fprintf(stderr, "Failed to launch hausdorffDistance kernel (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}


		//waits the processing
		hipDeviceSynchronize();

		//update the first image
		err = hipMemcpyAsync(d_img1, d_img1P, size, hipMemcpyDeviceToDevice);
		if (err != hipSuccess)
		{
			fprintf(stderr, "Failed to copy vector d_img1P to d_img1 from host to device (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}
		err = hipMemcpyAsync(d_img2, d_img2P, size, hipMemcpyDeviceToDevice);
		if (err != hipSuccess)
		{
			fprintf(stderr, "Failed to copy vector d_img2P to d_img2 from host to device (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}


		// Copy the device result vector in device memory to the host result vector
		// in host memory.
		//printf("Copy output data from the CUDA device to the host memory\n");
		err = hipMemcpyAsync(&h_grownEnough, d_grownEnough, sizeof(int), hipMemcpyDeviceToHost);
		if (err != hipSuccess)
		{
			fprintf(stderr, "Failed to copy d_grownEnough from device to host (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		hipDeviceSynchronize();

		distance++;
	}

	printf("Hausdorff distance: %d\n", distance);

	/*
	// Verify that the result vector is correct
	for (int i = 0; i < numElements; ++i)
	{
	if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
	{
	fprintf(stderr, "Result verification failed at element %d!\n", i);
	exit(EXIT_FAILURE);
	}
	}

	printf("Test PASSED\n");
	*/

	// Free device global memory
	err = hipFree(d_img1);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to free device vector img1 (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipFree(d_img2);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to free device vector img2 (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipFree(d_img1P);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to free device img1P (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipFree(d_img2P);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to free device img2P (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}


	err = hipFree(d_grownEnough);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to free device d_grownEnough (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipFree(d_structElement);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to free d_structElement (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}




	// Free host memory
	//free(h_distance);
	/*
	free(h_A);
	free(h_B);
	free(h_C);
	*/

	// Reset the device and exit
	// hipDeviceReset causes the driver to clean up all state. While
	// not mandatory in normal operation, it is good practice.  It is also
	// needed to ensure correct operation when the application is being
	// profiled. Calling hipDeviceReset causes all profile data to be
	// flushed before the application exits
	err = hipDeviceReset();

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	printf("Done\n");
	return 0;
}




